#include "hip/hip_runtime.h"
// Copyright 2015 Zhu.Jin Liang

#include <google/protobuf/text_format.h>
#include <google/protobuf/io/zero_copy_stream_impl.h>
#include <google/protobuf/io/coded_stream.h>

#include <cmath>

#include "caffe/common.hpp"
#include "caffe/util/util_img.hpp"
#include "caffe/util/util_pre_define.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/proto/caffe.pb.h"

namespace caffe {

template <typename Dtype>
__global__ void kernel_BiLinearResize(
		const int nthreads, 
		const Dtype* src_data, 
		const int src_height, 
		const int src_width,
		Dtype* dst_data, 
		const int dst_height, 
		const int dst_width, 
		const Dtype scale_h, 
		const Dtype scale_w)
{
	CUDA_KERNEL_LOOP(i, nthreads) {
		int dst_h = i /dst_width;
		Dtype fh = dst_h * scale_h;
		const int src_h = floor(fh);
		fh -= src_h;
		const Dtype w_h0 = std::abs(1.0f - fh);
		const Dtype w_h1 = std::abs(fh);

		const int dst_offset_1 =  dst_h * dst_width;
		const int src_offset_1 =  src_h * src_width;

		int dst_w = i %dst_width;
		Dtype fw = dst_w * scale_w;
		const int src_w = floor(fw);
		fw -= src_w;
		const Dtype w_w0 = std::abs(1.0f - fw);
		const Dtype w_w1 = std::abs(fw);

		const int dst_idx = dst_offset_1 + dst_w;
		dst_data[dst_idx] = 0;

		const int src_idx = src_offset_1 + src_w;

		dst_data[dst_idx] += (w_h0 * w_w0 * src_data[src_idx]);

		if (src_w + 1 < src_width) {
			dst_data[dst_idx] += 
				(w_h0 * w_w1 * src_data[src_idx + 1]);
		}

		if (src_h + 1 < src_height) {
			dst_data[dst_idx] += 
				(w_h1 * w_w0 * src_data[src_idx + src_width]);
		}

		if (src_w + 1 < src_width && src_h + 1 < src_height) {
			dst_data[dst_idx] += 
				(w_h1 * w_w1 * src_data[src_idx + src_width + 1]);
		}
	}
}

template <typename Dtype>
void BiLinearResizeMat_gpu(
		const Dtype* src, 
		const int src_height, 
		const int src_width,
		Dtype* dst, 
		const int dst_height, 
		const int dst_width)
{
	const Dtype scale_w = src_width / (Dtype)dst_width;
	const Dtype scale_h = src_height / (Dtype)dst_height;


	int loop_n = dst_height * dst_width;
	kernel_BiLinearResize<Dtype> 
		<<<CAFFE_GET_BLOCKS(loop_n), CAFFE_CUDA_NUM_THREADS >>>(
			loop_n,src, src_height, src_width, dst, 
			dst_height, dst_width, scale_h, scale_w);

	//CUDA_POST_KERNEL_CHECK;
}
template void BiLinearResizeMat_gpu(
		const float* src, 
		const int src_height, 
		const int src_width,
		float* dst, 
		const int dst_height, 
		const int dst_width);

template void BiLinearResizeMat_gpu(
		const double* src, 
		const int src_height, 
		const int src_width,
		double* dst, 
		const int dst_height, 
		const int dst_width);

template <typename Dtype>
void ResizeBlob_gpu(
		const Blob<Dtype>* src, 
		const int src_n, 
		const int src_c,
		Blob<Dtype>* dst, 
		const int dst_n, 
		const int dst_c,
		const bool data_or_diff /* true: data, false: diff */) 
{
	const int src_channels = src->channels();
	const int src_height = src->height();
	const int src_width = src->width();
	const int src_offset = 
			(src_n * src_channels + src_c) * 
				src_height * src_width;

	const int dst_channels = dst->channels();
	const int dst_height = dst->height();
	const int dst_width = dst->width();
	const int dst_offset = 
			(dst_n * dst_channels + dst_c) * 
				dst_height * dst_width;

	// const Dtype scale_w = src_width / (Dtype)dst_width;
	// const Dtype scale_h = src_height / (Dtype)dst_height;
	
	const Dtype* src_data = 
			data_or_diff ?
			&(src->gpu_data()[src_offset]) :
			&(src->gpu_diff()[src_offset]);

	Dtype* dst_data = 
			data_or_diff ? 
			&(dst->mutable_gpu_data()[dst_offset]) :
			&(dst->mutable_gpu_diff()[dst_offset]);

	BiLinearResizeMat_gpu(src_data,  src_height,  src_width,
			dst_data,  dst_height,  dst_width);
	CUDA_POST_KERNEL_CHECK;
}
template void ResizeBlob_gpu(
		const Blob<float>* src, 
		const int src_n, 
		const int src_c,
		Blob<float>* dst, 
		const int dst_n, 
		const int dst_c,
		const bool data_or_diff);
template void ResizeBlob_gpu(
		const Blob<double>* src, 
		const int src_n, 
		const int src_c,
		Blob<double>* dst, 
		const int dst_n, 
		const int dst_c,
		const bool data_or_diff);

template <typename Dtype>
__global__ void kernel_GetBiLinearResizeMatRules(
		const int nthreads,  
		const int src_height, 
		const int src_width,
		const int dst_height, 
		const int dst_width, 
		const Dtype scale_h, 
		const Dtype scale_w,
		Dtype* loc1, 
		Dtype* weight1, 
		Dtype* loc2, 
		Dtype* weight2,
		Dtype* loc3, 
		Dtype* weight3, 
		Dtype* loc4, 
		Dtype* weight4)
{
	CUDA_KERNEL_LOOP(index, nthreads)
	{
		int dst_h = index /dst_width;
		Dtype fh = dst_h * scale_h;
		const int src_h = floor(fh);
		fh -= src_h;
		const Dtype w_h0 = std::abs(1.0f - fh);
		const Dtype w_h1 = std::abs(fh);

		const int dst_offset_1 =  dst_h * dst_width;
		const int src_offset_1 =  src_h * src_width;

		int dst_w = index %dst_width;
		Dtype fw = dst_w * scale_w;
		const int src_w = floor(fw);
		fw -= src_w;
		const Dtype w_w0 = std::abs(1.0f - fw);
		const Dtype w_w1 = std::abs(fw);

		const int dst_idx = dst_offset_1 + dst_w;
		// dst_data[dst_idx] = 0;

		const int src_idx = src_offset_1 + src_w;

		loc1[dst_idx] = src_idx;
		weight1[dst_idx] = w_h0 * w_w0;

		if (src_w + 1 < src_width)
		{
			loc2[dst_idx] = src_idx + 1;
			weight2[dst_idx] = w_h0 * w_w1;
			// dst_data[dst_idx] += (w_h0 * w_w1 * src_data[src_idx + 1]);
		}

		if (src_h + 1 < src_height)
		{
			// dst_data[dst_idx] += (w_h1 * w_w0 * src_data[src_idx + src_width]);
			weight3[dst_idx] = w_h1 * w_w0;
			loc3[dst_idx] = src_idx + src_width;
		}

		if (src_w + 1 < src_width && src_h + 1 < src_height)
		{
			loc4[dst_idx] = src_idx + src_width + 1;
			weight4[dst_idx] = w_h1 * w_w1;
			// dst_data[dst_idx] += (w_h1 * w_w1 * src_data[src_idx + src_width + 1]);
		}
	}
}


template <typename Dtype>
__global__ void kernel_ResizeBlob(
		const int nthreads,
		const int num,
		const int channels, 
		const Dtype* src, 
		const int src_height, 
		const int src_width,
		Dtype* dst, 
		const int dst_height, 
		const int dst_width, 
		const Dtype scale_h, 
		const Dtype scale_w)
{
	CUDA_KERNEL_LOOP(index, nthreads) {
		int i = index %( dst_height * dst_width);
		int c = (index/(dst_height * dst_width))%channels;
		int n = (index/(dst_height * dst_width))/channels;
		int src_offset = (n * channels + c) * src_height * src_width;
		int dst_offset = (n * channels + c) * dst_height * dst_width;

		const Dtype* src_data = src+src_offset;
		Dtype* dst_data = dst+dst_offset;

		int dst_h = i /dst_width;
		Dtype fh = dst_h * scale_h;
		const int src_h = floor(fh);
		fh -= src_h;
		const Dtype w_h0 = std::abs(1.0f - fh);
		const Dtype w_h1 = std::abs(fh);

		const int dst_offset_1 =  dst_h * dst_width;
		const int src_offset_1 =  src_h * src_width;

		int dst_w = i %dst_width;
		Dtype fw = dst_w * scale_w;
		const int src_w = floor(fw);
		fw -= src_w;
		const Dtype w_w0 = std::abs(1.0f - fw);
		const Dtype w_w1 = std::abs(fw);

		const int dst_idx = dst_offset_1 + dst_w;
		dst_data[dst_idx] = 0;

		const int src_idx = src_offset_1 + src_w;

		dst_data[dst_idx] += 
				(w_h0 * w_w0 * src_data[src_idx]);

		if (src_w + 1 < src_width) {
			dst_data[dst_idx] += 
				(w_h0 * w_w1 * src_data[src_idx + 1]);
		}

		if (src_h + 1 < src_height) {
			dst_data[dst_idx] += 
				(w_h1 * w_w0 * src_data[src_idx + src_width]);
		}

		if (src_w + 1 < src_width && src_h + 1 < src_height) {
			dst_data[dst_idx] += 
				(w_h1 * w_w1 * src_data[src_idx + src_width + 1]);
		}
	}
}

template <typename Dtype>
void ResizeBlob_gpu(
		const Blob<Dtype>* src, 
		Blob<Dtype>* dst, 
		/* true: data, false: diff */
		const bool data_or_diff) 
{
	CHECK(src->num() == dst->num())
			<<"src->num() == dst->num()";
	CHECK(src->channels() == dst->channels())
			<< "src->channels() == dst->channels()";

	const int src_num = src->num();
	const int src_channels = src->channels();
	const int src_height = src->height();
	const int src_width = src->width();


	const int dst_channels = dst->channels();
	const int dst_height = dst->height();
	const int dst_width = dst->width();


	const Dtype scale_w = src_width / (Dtype)dst_width;
	const Dtype scale_h = src_height / (Dtype)dst_height;
	int loop_n = dst_height * dst_width*dst_channels*src_num;

	const Dtype* src_data = 
			data_or_diff ? 
			src->gpu_data() :
			src->gpu_diff();
	Dtype* dst_data = 
			data_or_diff ? 
			dst->mutable_gpu_data() :
			dst->mutable_gpu_diff();

	kernel_ResizeBlob<Dtype> <<<CAFFE_GET_BLOCKS(loop_n), 
		CAFFE_CUDA_NUM_THREADS >>>
			(loop_n,src_num,src_channels,
				src_data, src_height,src_width,
				dst_data, dst_height, dst_width,
				scale_h,scale_w);
	CUDA_POST_KERNEL_CHECK;
}
template void ResizeBlob_gpu(
		const Blob<float>* src,
		Blob<float>* dst,
		const bool data_or_diff);
template void ResizeBlob_gpu(
		const Blob<double>* src,
		Blob<double>* dst,
		const bool data_or_diff);

template <typename Dtype>
void ResizeBlob_Data_gpu(
		const Blob<Dtype>* src, 
		Blob<Dtype>* dst)
{
	const bool data_or_diff = true;
	ResizeBlob_gpu(src, dst, data_or_diff);
}
template void ResizeBlob_Data_gpu(
		const Blob<float>* src,
		Blob<float>* dst);
template void ResizeBlob_Data_gpu(
		const Blob<double>* src,
		Blob<double>* dst);

template <typename Dtype>
void ResizeBlob_Diff_gpu(
		const Blob<Dtype>* src, 
		Blob<Dtype>* dst)
{
	const bool data_or_diff = false;
	ResizeBlob_gpu(src, dst, data_or_diff);
}
template void ResizeBlob_Diff_gpu(
		const Blob<float>* src,
		Blob<float>* dst);
template void ResizeBlob_Diff_gpu(
		const Blob<double>* src,
		Blob<double>* dst);

template <typename Dtype>
void GetBiLinearResizeMatRules_gpu( 
		const int src_height, 
		const int src_width,
		const int dst_height, 
		const int dst_width,
		Dtype* loc1, 
		Dtype* weight1, 
		Dtype* loc2, 
		Dtype* weight2,
		Dtype* loc3, 
		Dtype* weight3, 
		Dtype* loc4, 
		Dtype* weight4)
{
	const Dtype scale_w = src_width / (Dtype)dst_width;
	const Dtype scale_h = src_height / (Dtype)dst_height;


	int loop_n = dst_height * dst_width;
	caffe::caffe_gpu_set(loop_n,(Dtype)0,loc1);
	caffe::caffe_gpu_set(loop_n,(Dtype)0,loc2);
	caffe::caffe_gpu_set(loop_n,(Dtype)0,loc4);
	caffe::caffe_gpu_set(loop_n,(Dtype)0,loc3);

	caffe::caffe_gpu_set(loop_n,(Dtype)0,weight1);
	caffe::caffe_gpu_set(loop_n,(Dtype)0,weight2);
	caffe::caffe_gpu_set(loop_n,(Dtype)0,weight3);
	caffe::caffe_gpu_set(loop_n,(Dtype)0,weight4);
	kernel_GetBiLinearResizeMatRules<Dtype> 
		<<<CAFFE_GET_BLOCKS(loop_n), CAFFE_CUDA_NUM_THREADS >>>(
			loop_n,  src_height,  src_width,
			dst_height, dst_width, scale_h, scale_w,
			loc1,  weight1,  loc2,  weight2,
			loc3,  weight3,   loc4,   weight4);
	CUDA_POST_KERNEL_CHECK;
}
template void GetBiLinearResizeMatRules_gpu(  
		const int src_height, 
		const int src_width,
		const int dst_height, 
		const int dst_width,
		float* loc1, 
		float* weight1, 
		float* loc2, 
		float* weight2,
		float* loc3, 
		float* weight3, 
		float* loc4, 
		float* weight4);
template void GetBiLinearResizeMatRules_gpu(  
		const int src_height, 
		const int src_width,
		const int dst_height, 
		const int dst_width,
		double* loc1, 
		double* weight1, 
		double* loc2, 
		double* weight2,
		double* loc3, 
		double* weight3, 
		double* loc4, 
		double* weight4);

template <typename Dtype>
void ResizeBlob_gpu(
		const Blob<Dtype>* src,
		Blob<Dtype>* dst,
		Blob<Dtype>* loc1, 
		Blob<Dtype>* loc2, 
		Blob<Dtype>* loc3, 
		Blob<Dtype>* loc4)
{
	CHECK(src->num() == dst->num())
			<<"src->num() == dst->num()";
	CHECK(src->channels() == dst->channels())
			<< "src->channels() == dst->channels()";

	GetBiLinearResizeMatRules_gpu(
			src->height(),
			src->width(),
			dst->height(), 
			dst->width(),
			loc1->mutable_gpu_data(), 
			loc1->mutable_gpu_diff(), 
			loc2->mutable_gpu_data(), 
			loc2->mutable_gpu_diff(),
			loc3->mutable_gpu_data(), 
			loc3->mutable_gpu_diff(), 
			loc4->mutable_gpu_data(), 
			loc4->mutable_gpu_diff());

	for(int n=0;n< src->num();++n)
	{
		for(int c=0; c < src->channels() ; ++c)
		{
			ResizeBlob_gpu(src,n,c,dst,n,c);
		}
	}
}
template void ResizeBlob_gpu(
		const Blob<float>* 
		src,Blob<float>* dst,
		Blob<float>* loc1, 
		Blob<float>* loc2, 
		Blob<float>* loc3, 
		Blob<float>* loc4);
template void ResizeBlob_gpu(
		const Blob<double>* src,
		Blob<double>* dst,
		Blob<double>* loc1, 
		Blob<double>* loc2, 
		Blob<double>* loc3, 
		Blob<double>* loc4);

template <typename Dtype>
__global__ void affine_warp_blob_kernel(const int nthreads,
		const Dtype* src_data, const int src_step,
		Dtype* dst_data, const int dst_step, const int dst_channels,
		const Dtype* weights, const int* locs,
		const int weight_num, const int weight_channels,
		const int weight_height, const int weight_width) {

  CUDA_KERNEL_LOOP(index, nthreads) {

  	// 把num跟channels合成一个维度
  	// 算出当前是第几个height * width
  	// 以及在该height * width下的位置
  	const int i = index / dst_step;
  	const int j = index % dst_step;

  	// 算出是第几个样本
  	const int n = i / dst_channels;
  	// 求出是第几个channels
  	const int c = i % dst_channels;
  	// 算出该channels下对应的x/y映射的offset
  	const int weight_n = n % weight_num;
  	const int weight_c = c / (dst_channels / weight_channels);

  	const int in_offset = i * src_step;
  	const int weights_offset = ((weight_n * weight_channels + weight_c) * weight_height + j) * weight_width;

  	dst_data[index] = 0;
  	for (int k = 0; k < weight_width; ++k) {
  		dst_data[index] += (weights[weights_offset + k] * src_data[in_offset + locs[weights_offset + k]]);
  	}
  }
}

template <typename Dtype>
void AffineWarpBlob_gpu(const Blob<Dtype>* src, Blob<Dtype>* dst,
		const vector<pair<Dtype, Dtype> >& coefs,
		const int coord_maps_count, const int num) {

	Blob<Dtype> weights;
	Blob<int> locs;
	GetResizeRules(src->height(), src->width(),
			dst->height(), dst->width(),
			&weights, &locs, coefs, coord_maps_count, num);

	AffineWarpBlob_gpu(src, dst, &weights, &locs);
}

template void AffineWarpBlob_gpu(const Blob<float>* src, Blob<float>* dst,
		const vector<pair<float, float> >& coefs,
		const int coord_maps_count, const int num);
template void AffineWarpBlob_gpu(const Blob<double>* src, Blob<double>* dst,
		const vector<pair<double, double> >& coefs,
		const int coord_maps_count, const int num);

template <typename Dtype>
void AffineWarpBlob_gpu(const Blob<Dtype>* src, Blob<Dtype>* dst,
		const Blob<Dtype>* weights, const Blob<int>* locs) {

	const Dtype* src_data = src->gpu_data();
	const int src_step = src->height() * src->width();

	Dtype* dst_data = dst->mutable_gpu_data();
	const int dst_step = dst->height() * dst->width();

	const int count = dst->count();

	affine_warp_blob_kernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
			count,
			src_data, src_step,
			dst_data, dst_step, dst->channels(),
			weights->gpu_data(), locs->gpu_data(),
			weights->num(), weights->channels(),
			weights->height(), weights->width());
	CUDA_POST_KERNEL_CHECK;

}

template void AffineWarpBlob_gpu(const Blob<float>* src, Blob<float>* dst,
		const Blob<float>* weights, const Blob<int>* locs);
template void AffineWarpBlob_gpu(const Blob<double>* src, Blob<double>* dst,
		const Blob<double>* weights, const Blob<int>* locs);

} // namespace caffe
