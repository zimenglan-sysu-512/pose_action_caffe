#include "hip/hip_runtime.h"
// Copyright 2015 DDK (dongdk.sysu@foxmail.com)

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/pose_estimation_layers.hpp"
#include "caffe/layer.hpp"
#include "caffe/common.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

using std::min;
using std::max;

template <typename Dtype>
void CoordsToBboxesMasksLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) 
{
  // Get coordinates
  this->Forward_cpu(bottom, top);
}

template <typename Dtype>
void CoordsToBboxesMasksLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) 
{
  const Dtype Zero = Dtype(0);
  CHECK_EQ(propagate_down.size(), bottom.size());

  for (int i = 0; i < propagate_down.size(); ++i) {
    if (propagate_down[i]) { 
      // NOT_IMPLEMENTED; 
      caffe_set(bottom[i]->count(), Zero, bottom[i]->mutable_gpu_diff());
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CoordsToBboxesMasksLayer);

}  // namespace caffe